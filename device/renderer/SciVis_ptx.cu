#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2022 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice,
 * this list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
 * LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
 * CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
 * SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
 * INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
 * CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
 * ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 * POSSIBILITY OF SUCH DAMAGE.
 */

#include "gpu/shading_api.h"

namespace visrtx {

enum class RayType
{
  PRIMARY = 0,
  SHADOW = 1
};

struct RayAttenuation
{
  const Ray *ray{nullptr};
  float attenuation{0.f};
};

DECLARE_FRAME_DATA(frameData)

// Helper functions ///////////////////////////////////////////////////////////

RT_FUNCTION bool isOccluded(ScreenSample &ss, Ray r)
{
  uint32_t o = 0;
  intersectSurface(
      ss, r, RayType::SHADOW, &o, OPTIX_RAY_FLAG_DISABLE_CLOSESTHIT);
  return static_cast<bool>(o);
}

RT_FUNCTION float computeAO(
    ScreenSample &ss, const Ray &primaryRay, const Hit &currentHit)
{
  int hits = 0;

  const int numSamples = frameData.renderer.params.scivis.aoSamples;
  for (int i = 0; i < numSamples; i++) {
    Ray aoRay;
    aoRay.org = currentHit.hitpoint + (currentHit.epsilon * currentHit.Ng);
    aoRay.dir = randomDir(ss.rs, currentHit.Ns);
    if (isOccluded(ss, aoRay))
      hits++;
  }

  return 1.f - hits / float(numSamples);
}

RT_FUNCTION float attenuation(ScreenSample &ss, Ray r)
{
  RayAttenuation ra;
  ra.ray = &r;
  intersectVolume(
      ss, r, RayType::SHADOW, &ra, OPTIX_RAY_FLAG_DISABLE_CLOSESTHIT);
  return ra.attenuation;
}

RT_FUNCTION vec3 computeLightConrib(ScreenSample &ss, const SurfaceHit &hit)
{
  const auto &rendererParams = frameData.renderer.params.scivis;

  auto &world = frameData.world;

  const vec3 shadePoint = hit.hitpoint + (hit.epsilon * hit.Ng);

  vec3 contrib(0.f);
  for (size_t i = 0; i < world.numLightInstances; i++) {
    auto *inst = world.lightInstances + i;
    if (!inst)
      continue;

    for (size_t l = 0; l < inst->numLights; l++) {
      auto ls = sampleLight(ss, hit, inst->indices[l]);
      Ray r;
      r.org = shadePoint;
      r.dir = ls.dir;
      r.t.upper = ls.dist;
      if (!isOccluded(ss, r)) {
        contrib += ls.radiance * dot(ls.dir, hit.Ns)
            * rendererParams.lightFalloff * (1.f - attenuation(ss, r));
      }
    }
  }
  return contrib;
}

// OptiX programs /////////////////////////////////////////////////////////////

RT_PROGRAM void __closesthit__shadow()
{
  // no-op
}

RT_PROGRAM void __anyhit__shadow()
{
  if (ray::isIntersectingSurfaces()) {
    SurfaceHit hit;
    ray::populateSurfaceHit(hit);
    const auto &material = *hit.material;
    const auto mat_baseColor =
        getMaterialParameter(frameData, material.baseColor, hit);
    const auto mat_opacity =
        getMaterialParameter(frameData, material.opacity, hit);
    if (mat_opacity >= 0.99f) {
      auto &occluded = ray::rayData<uint32_t>();
      occluded = true;
      optixTerminateRay();
    } else
      optixIgnoreIntersection();
  } else {
    auto &ra = ray::rayData<RayAttenuation>();
    VolumeHit hit;
    ray::populateVolumeHit(hit);
    rayMarchVolume(ray::screenSample(), hit, ra.attenuation);
    if (ra.attenuation < 0.99f)
      optixIgnoreIntersection();
  }
}

RT_PROGRAM void __closesthit__primary()
{
  ray::populateHit();
}

RT_PROGRAM void __miss__()
{
  // TODO
}

RT_PROGRAM void __raygen__()
{
  const auto &rendererParams = frameData.renderer.params.scivis;

  /////////////////////////////////////////////////////////////////////////////
  // TODO: clean this up! need to split out Ray/RNG, don't need screen samples
  auto ss = createScreenSample(frameData);
  if (pixelOutOfFrame(ss.pixel, frameData.fb))
    return;
  auto ray = makePrimaryRay(ss);
  float tmax = ray.t.upper;
  /////////////////////////////////////////////////////////////////////////////

  SurfaceHit surfaceHit;
  VolumeHit volumeHit;
  vec3 outputColor(0.f);
  vec3 outputNormal = ray.dir;
  float outputOpacity = 0.f;
  float depth = 1e30f;
  bool firstHit = true;

  while (outputOpacity < 0.99f) {
    ray.t.upper = tmax;
    surfaceHit.foundHit = false;
    intersectSurface(ss, ray, RayType::PRIMARY, &surfaceHit);

    vec3 color(0.f);
    float opacity = 0.f;

    if (surfaceHit.foundHit) {
      depth = min(depth,
          rayMarchAllVolumes(
              ss, ray, RayType::PRIMARY, surfaceHit.t, color, opacity));

      if (firstHit) {
        outputNormal = surfaceHit.Ng;
        depth = min(depth, surfaceHit.t);
        firstHit = false;
      }

      const auto &material = *surfaceHit.material;
      const auto mat_baseColor =
          getMaterialParameter(frameData, material.baseColor, surfaceHit);
      const auto mat_opacity =
          getMaterialParameter(frameData, material.opacity, surfaceHit);

      const float aoFactor = rendererParams.aoSamples > 0
          ? computeAO(ss, ray, surfaceHit) * rendererParams.aoIntensity
          : 0.f;

      accumulateValue(color,
          (mat_baseColor * computeLightConrib(ss, surfaceHit))
              + (rendererParams.aoColor * aoFactor
                  * rendererParams.lightFalloff),
          opacity);
      accumulateValue(opacity, mat_opacity, opacity);

      color *= opacity;
      accumulateValue(outputColor, color, outputOpacity);
      accumulateValue(outputOpacity, opacity, outputOpacity);

      ray.t.lower = surfaceHit.t + surfaceHit.epsilon;
    } else {
      const auto bgColor = vec3(frameData.renderer.bgColor);
      const auto bgOpacity = frameData.renderer.bgColor.w;

      const float volumeDepth = rayMarchAllVolumes(
          ss, ray, RayType::PRIMARY, ray.t.upper, color, opacity);

      if (firstHit)
        depth = min(depth, volumeDepth);

      color *= opacity;
      accumulateValue(color, bgColor, opacity);
      accumulateValue(opacity, bgOpacity, opacity);
      accumulateValue(outputColor, color, outputOpacity);
      accumulateValue(outputOpacity, opacity, outputOpacity);
      break;
    }
  }

  accumResults(frameData.fb,
      ss.pixel,
      vec4(outputColor, outputOpacity),
      depth,
      outputColor,
      outputNormal);
}

} // namespace visrtx
