#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2025 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice,
 * this list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
 * LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
 * CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
 * SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
 * INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
 * CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
 * ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 * POSSIBILITY OF SUCH DAMAGE.
 */

#include <hip/hip_runtime_api.h>
#include "UniformGrid.h"
#include "gpu/gpu_math.h"
#include "gpu/gpu_objects.h"
#include "gpu/sampleSpatialField.h"
#include "gpu/uniformGrid.h"
#ifdef __CUDA_ARCH__
#include "gpu/gpu_util.h"
#endif

namespace visrtx {

__global__ void invalidateRangesGPU(box1 *valueRanges, const ivec3 dims)
{
  size_t threadID = blockIdx.x * size_t(blockDim.x) + threadIdx.x;

  if (threadID >= dims.x * size_t(dims.y) * dims.z)
    return;

  valueRanges[threadID].lower = +1e30f;
  valueRanges[threadID].upper = -1e30f;
}

__global__ void computeMaxOpacitiesGPU(float *maxOpacities,
    const box1 *valueRanges,
    hipTextureObject_t colorMap,
    size_t numMCs,
    size_t numColors,
    box1 xfRange)
{
  size_t threadID = blockIdx.x * size_t(blockDim.x) + threadIdx.x;

  if (threadID >= numMCs)
    return;

  box1 valueRange = valueRanges[threadID];

  if (valueRange.upper < valueRange.lower) {
    maxOpacities[threadID] = 0.f;
    return;
  }

  valueRange.lower -= xfRange.lower;
  valueRange.lower /= xfRange.upper - xfRange.lower;
  valueRange.upper -= xfRange.lower;
  valueRange.upper /= xfRange.upper - xfRange.lower;

  int lo = glm::clamp(
      int(valueRange.lower * (numColors - 1)), 0, int(numColors - 1));
  int hi = glm::clamp(
      int(valueRange.upper * (numColors - 1)) + 1, 0, int(numColors - 1));

  float maxOpacity = 0.f;
  for (int i = lo; i <= hi; ++i) {
    float tc = (i + .5f) / numColors;
    maxOpacity = fmaxf(maxOpacity, tex1D<float4>(colorMap, tc).w);
  }
  maxOpacities[threadID] = maxOpacity;
}

template <typename Sampler>
__global__ void buildGridGPU(box1 *valueRanges,
    ivec3 dims,
    box3 worldBounds,
    const SpatialFieldGPUData *sfgd)
{
  Sampler sampler(*sfgd);

  size_t threadID = blockIdx.x * size_t(blockDim.x) + threadIdx.x;

  size_t numVoxels = (dims.x - 1) * size_t(dims.y - 1) * (dims.z - 1);

  if (threadID >= numVoxels)
    return;

  ivec3 voxelID(threadID % (dims.x - 1),
      threadID / (dims.x - 1) % (dims.y - 1),
      threadID / ((dims.x - 1) * (dims.y - 1)));

  vec3 worldExtend = size(worldBounds);
  vec3 voxelExtend = worldExtend / vec3(dims - 1);
  box3 voxelBounds(worldBounds.lower + vec3(voxelID) * voxelExtend,
      worldBounds.lower + vec3(voxelID) * voxelExtend + voxelExtend);

  // compute the max value of all the cells that can
  // overlap this voxel; splat out the _max_ over the
  // overlapping MCs. (that's essentially a box filter)
  vec3 tcs[8] = {(vec3(voxelID) + vec3(-.5f, -.5f, -.5f)) / vec3(dims),
      (vec3(voxelID) + vec3(+.5f, -.5f, -.5f)) / vec3(dims),
      (vec3(voxelID) + vec3(+.5f, +.5f, -.5f)) / vec3(dims),
      (vec3(voxelID) + vec3(-.5f, +.5f, -.5f)) / vec3(dims),
      (vec3(voxelID) + vec3(-.5f, -.5f, +.5f)) / vec3(dims),
      (vec3(voxelID) + vec3(+.5f, -.5f, +.5f)) / vec3(dims),
      (vec3(voxelID) + vec3(+.5f, +.5f, +.5f)) / vec3(dims),
      (vec3(voxelID) + vec3(-.5f, +.5f, +.5f)) / vec3(dims)};

  float voxelValue = -1e30f;
  for (int i = 0; i < 8; ++i) {
    float retval = sampler(vec3(tcs[i].x, tcs[i].y, tcs[i].z));
    voxelValue = fmaxf(voxelValue, retval);
  }

  // find out which MCs we overlap and splat the value out
  // on the respective ranges
  const ivec3 loMC = projectOnGrid(voxelBounds.lower, dims, worldBounds);
  const ivec3 upMC = projectOnGrid(voxelBounds.upper, dims, worldBounds);

  for (int mcz = loMC.z; mcz <= upMC.z; ++mcz) {
    for (int mcy = loMC.y; mcy <= upMC.y; ++mcy) {
      for (int mcx = loMC.x; mcx <= upMC.x; ++mcx) {
        const ivec3 mcID(mcx, mcy, mcz);
#ifdef __CUDA_ARCH__
        atomicMinf(&valueRanges[linearIndex(mcID, dims)].lower, voxelValue);
        atomicMaxf(&valueRanges[linearIndex(mcID, dims)].upper, voxelValue);
#endif
      }
    }
  }
}

void UniformGrid::init(ivec3 dims, box3 worldBounds)
{
  m_dims = ivec3(iDivUp(dims.x, 16), iDivUp(dims.y, 16), iDivUp(dims.z, 16));
  m_worldBounds = worldBounds;

  size_t numMCs = m_dims.x * size_t(m_dims.y) * m_dims.z;

  hipFree(m_valueRanges);
  hipFree(m_maxOpacities);

  hipMalloc(&m_valueRanges, numMCs * sizeof(box1));
  hipMalloc(&m_maxOpacities, numMCs * sizeof(float));

  size_t numThreads = 1024;
  invalidateRangesGPU<<<(uint32_t)iDivUp(numMCs, numThreads),
      (uint32_t)numThreads>>>(m_valueRanges, m_dims);
}

void UniformGrid::buildGrid(const SpatialFieldGPUData &sfgd)
{
  size_t numVoxels = (m_dims.x - 1) * size_t(m_dims.y - 1) * (m_dims.z - 1);
  size_t numThreads = 1024;

  // We ned to get the spatialfield gpu data upload, but we don't get
  // to access the framedata store.
  // Let's do a temporary upload so we can do the job.
  SpatialFieldGPUData *sfgdDevice = {};
  hipMalloc(&sfgdDevice, sizeof(sfgd));
  hipMemcpy(sfgdDevice, &sfgd, sizeof(sfgd), hipMemcpyHostToDevice);

  switch (sfgd.type) {
  case SpatialFieldType::STRUCTURED_REGULAR: {
    buildGridGPU<SpatialFieldSampler<hipTextureObject_t>>
        <<<iDivUp(numVoxels, numThreads), numThreads>>>(
            m_valueRanges, m_dims, m_worldBounds, sfgdDevice);
    break;
  }
  case SpatialFieldType::NANOVDB_REGULAR: {
    switch (sfgd.data.nvdbRegular.gridType) {
    case nanovdb::GridType::Fp4: {
      buildGridGPU<NvdbSpatialFieldSampler<nanovdb::Fp4>>
          <<<iDivUp(numVoxels, numThreads), numThreads>>>(
              m_valueRanges, m_dims, m_worldBounds, sfgdDevice);
      break;
    }
    case nanovdb::GridType::Fp8: {
      buildGridGPU<NvdbSpatialFieldSampler<nanovdb::Fp8>>
          <<<iDivUp(numVoxels, numThreads), numThreads>>>(
              m_valueRanges, m_dims, m_worldBounds, sfgdDevice);
      break;
    }
    case nanovdb::GridType::Fp16: {
      buildGridGPU<NvdbSpatialFieldSampler<nanovdb::Fp16>>
          <<<iDivUp(numVoxels, numThreads), numThreads>>>(
              m_valueRanges, m_dims, m_worldBounds, sfgdDevice);
      break;
    }
    case nanovdb::GridType::FpN: {
      buildGridGPU<NvdbSpatialFieldSampler<nanovdb::FpN>>
          <<<iDivUp(numVoxels, numThreads), numThreads>>>(
              m_valueRanges, m_dims, m_worldBounds, sfgdDevice);
      break;
    }
    case nanovdb::GridType::Float: {
      buildGridGPU<NvdbSpatialFieldSampler<float>>
          <<<iDivUp(numVoxels, numThreads), numThreads>>>(
              m_valueRanges, m_dims, m_worldBounds, sfgdDevice);
      break;
    }
    default:
      break;
    }
    break;
  }
  }

  hipFree(sfgdDevice);
}

void UniformGrid::cleanup()
{
  hipFree(m_valueRanges);
  hipFree(m_maxOpacities);

  m_valueRanges = nullptr;
  m_maxOpacities = nullptr;
}

UniformGridData UniformGrid::gpuData() const
{
  UniformGridData grid;
  grid.dims = m_dims;
  grid.worldBounds = m_worldBounds;
  grid.valueRanges = m_valueRanges;
  grid.maxOpacities = m_maxOpacities;
  return grid;
}

void UniformGrid::computeMaxOpacities(
    hipStream_t stream, hipTextureObject_t cm, size_t cmSize, box1 cmRange)
{
  size_t numMCs = m_dims.x * size_t(m_dims.y) * m_dims.z;

  size_t numThreads = 1024;
  computeMaxOpacitiesGPU<<<iDivUp(numMCs, numThreads), numThreads, 0, stream>>>(
      m_maxOpacities, m_valueRanges, cm, numMCs, cmSize, cmRange);
}

} // namespace visrtx
