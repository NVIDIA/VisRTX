#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2024 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice,
 * this list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
 * LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
 * CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
 * SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
 * INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
 * CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
 * ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 * POSSIBILITY OF SUCH DAMAGE.
 */

#include "DebugMethod.h"
#include "gpu/shading_api.h"

namespace visrtx {

enum class RayType
{
  DEBUG
};

struct SurfaceRayData : public SurfaceHit
{
  vec3 outColor{0.f};
};

struct VolumeRayData : public VolumeHit
{
  vec3 outColor{0.f};
};

DECLARE_FRAME_DATA(frameData)

VISRTX_DEVICE void handleSurfaceHit()
{
  auto &rd = ray::rayData<SurfaceRayData>();
  ray::populateSurfaceHit(rd);

  const auto method =
      static_cast<DebugMethod>(frameData.renderer.params.debug.method);

  switch (method) {
  case DebugMethod::PRIM_ID:
    rd.outColor = makeRandomColor(ray::primID());
    break;
  case DebugMethod::GEOM_ID:
    rd.outColor = makeRandomColor(ray::objID());
    break;
  case DebugMethod::INST_ID:
    rd.outColor = makeRandomColor(ray::instID());
    break;
  case DebugMethod::RAY_UVW:
    rd.outColor = ray::uvw(rd.geometry->type);
    break;
  case DebugMethod::IS_TRIANGLE:
    rd.outColor = boolColor(rd.geometry->type == GeometryType::TRIANGLE);
    break;
  case DebugMethod::IS_VOLUME:
    rd.outColor = boolColor(false);
    break;
  case DebugMethod::BACKFACE:
    rd.outColor = boolColor(optixIsFrontFaceHit());
    break;
  case DebugMethod::NG:
    rd.outColor = rd.Ng;
    break;
  case DebugMethod::NG_ABS:
    rd.outColor = abs(rd.Ng);
    break;
  case DebugMethod::NS:
    rd.outColor = rd.Ns;
    break;
  case DebugMethod::NS_ABS:
    rd.outColor = abs(rd.Ns);
    break;
  case DebugMethod::HAS_MATERIAL:
    rd.outColor = boolColor(rd.material);
    break;
  case DebugMethod::GEOMETRY_ATTRIBUTE_0:
    rd.outColor = readAttributeValue(0, rd);
    break;
  case DebugMethod::GEOMETRY_ATTRIBUTE_1:
    rd.outColor = readAttributeValue(1, rd);
    break;
  case DebugMethod::GEOMETRY_ATTRIBUTE_2:
    rd.outColor = readAttributeValue(2, rd);
    break;
  case DebugMethod::GEOMETRY_ATTRIBUTE_3:
    rd.outColor = readAttributeValue(3, rd);
    break;
  case DebugMethod::GEOMETRY_ATTRIBUTE_COLOR:
    rd.outColor = readAttributeValue(4, rd);
    break;
  default:
    rd.outColor = vec3(1.f);
    break;
  }

  const auto c = rd.outColor * glm::abs(glm::dot(ray::direction(), rd.Ns));
  rd.outColor = glm::mix(rd.outColor, c, 0.5f);
}

VISRTX_DEVICE void handleVolumeHit()
{
  auto &rd = ray::rayData<VolumeRayData>();
  ray::populateVolumeHit(rd);

  const auto method =
      static_cast<DebugMethod>(frameData.renderer.params.debug.method);

  switch (method) {
  case DebugMethod::PRIM_ID:
    rd.outColor = makeRandomColor(ray::primID());
    break;
  case DebugMethod::GEOM_ID:
    rd.outColor = makeRandomColor(ray::objID());
    break;
  case DebugMethod::INST_ID:
    rd.outColor = makeRandomColor(ray::instID());
    break;
  case DebugMethod::IS_TRIANGLE:
    rd.outColor = boolColor(false);
    break;
  case DebugMethod::IS_VOLUME:
    rd.outColor = boolColor(true);
    break;
  case DebugMethod::BACKFACE:
    rd.outColor = boolColor(optixIsFrontFaceHit());
    break;
  default:
    rd.outColor = vec3(1.f);
    break;
  }
}

VISRTX_GLOBAL void __closesthit__()
{
  if (ray::isIntersectingSurfaces())
    handleSurfaceHit();
  else
    handleVolumeHit();
}

VISRTX_GLOBAL void __miss__()
{
  // no-op
}

VISRTX_GLOBAL void __raygen__()
{
  auto ss = createScreenSample(frameData);
  if (pixelOutOfFrame(ss.pixel, frameData.fb))
    return;
  auto ray = makePrimaryRay(ss, true /*pixel centered*/);

  auto color = vec3(getBackground(frameData.renderer, ss.screen));
  auto depth = ray.t.upper;
  auto normal = ray.dir;
  uint32_t primID = ~0u;
  uint32_t objID = ~0u;
  uint32_t instID = ~0u;

  SurfaceRayData srd{};
  intersectSurface(ss, ray, RayType::DEBUG, &srd);

  VolumeRayData vrd{};
  intersectVolume(ss, ray, RayType::DEBUG, &vrd);

  if (srd.foundHit && vrd.foundHit) {
    const bool volumeFirst = vrd.localRay.t.lower < srd.t;
    if (volumeFirst) {
      color = vrd.outColor;
      depth = vrd.localRay.t.lower;
      normal = -ray.dir;
      primID = 0;
      objID = vrd.volumeData->id;
      instID = vrd.instID;
    } else {
      color = srd.outColor;
      depth = srd.t;
      normal = srd.Ng;
      primID = srd.primID;
      objID = srd.objID;
      instID = srd.instID;
    }
  } else if (srd.foundHit) {
    color = srd.outColor;
    depth = srd.t;
    normal = srd.Ng;
    primID = srd.primID;
    objID = srd.objID;
    instID = srd.instID;
  } else if (vrd.foundHit) {
    color = vrd.outColor;
    depth = vrd.localRay.t.lower;
    normal = -ray.dir;
    primID = 0;
    objID = vrd.volumeData->id;
    instID = vrd.instID;
  }

  accumResults(frameData.fb,
      ss.pixel,
      vec4(color, 1.f),
      depth,
      color,
      normal,
      primID,
      objID,
      instID);
}

} // namespace visrtx
