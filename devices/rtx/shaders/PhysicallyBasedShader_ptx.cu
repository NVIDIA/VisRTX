#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2025 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice,
 * this list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
 * LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
 * CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
 * SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
 * INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
 * CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
 * ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 * POSSIBILITY OF SUCH DAMAGE.
 */

#include "gpu/gpu_decl.h"
#include "gpu/gpu_objects.h"
#include "gpu/intersectRay.h"
#include "gpu/sampleLight.h"
#include "gpu/shadingState.h"
#include "gpu/shading_api.h"

using namespace visrtx;

VISRTX_CALLABLE void __direct_callable__init(
    PhysicallyBasedShadingState *shadingState,
    const FrameGPUData *fd,
    const SurfaceHit *hit,
    const MaterialGPUData::PhysicallyBased *md)
{
  vec4 color = getMaterialParameter(*fd, md->baseColor, *hit);
  float opacity = getMaterialParameter(*fd, md->opacity, *hit).x;

  shadingState->baseColor = vec3(color);
  shadingState->opacity =
      adjustedMaterialOpacity(color.w * opacity, md->alphaMode, md->cutoff);
  shadingState->ior = md->ior;
  shadingState->metallic = getMaterialParameter(*fd, md->metallic, *hit).x;
  shadingState->roughness = getMaterialParameter(*fd, md->roughness, *hit).x;
}

VISRTX_CALLABLE NextRay __direct_callable__nextRay(
    const PhysicallyBasedShadingState *shadingState,
    const Ray *ray,
    const ScreenSample *ss)
{
  return NextRay{vec3(0.0f, 0.0f, 0.0f), vec3(0.0f, 0.0f, 0.0f)};
}

VISRTX_CALLABLE
vec3 __direct_callable__evaluateTint(
    const PhysicallyBasedShadingState *shadingState)
{
  return shadingState->baseColor;
}

VISRTX_CALLABLE
float __direct_callable__evaluateOpacity(
    const PhysicallyBasedShadingState *shadingState)
{
  return shadingState->opacity;
}

// Signature must match the call inside shaderPhysicallyBasedSurface in
// PhysicallyBasedShader.cuh.
VISRTX_CALLABLE vec3 __direct_callable__shadeSurface(
    const PhysicallyBasedShadingState *shadingState,
    const SurfaceHit *hit,
    const LightSample *lightSample,
    const vec3 *outgoingDir)
{
  const vec3 H = normalize(lightSample->dir + *outgoingDir);
  const float NdotH = dot(hit->Ns, H);
  const float NdotL = dot(hit->Ns, lightSample->dir);
  const float NdotV = dot(hit->Ns, *outgoingDir);
  const float VdotH = dot(*outgoingDir, H);
  const float LdotH = dot(lightSample->dir, H);

  // Fresnel
  const vec3 f0 = glm::mix(
      vec3(pow2((1.f - shadingState->ior) / (1.f + shadingState->ior))),
      shadingState->baseColor,
      shadingState->metallic);
  const vec3 F = f0 + (vec3(1.f) - f0) * pow5(1.f - fabsf(VdotH));

  // Metallic materials don't reflect diffusely:
  const vec3 diffuseColor =
      glm::mix(shadingState->baseColor, vec3(0.f), shadingState->metallic);

  const vec3 diffuseBRDF =
      (vec3(1.f) - F) * float(M_1_PI) * diffuseColor * fmaxf(0.f, NdotL);

  // Alpha
  const float alpha = pow2(shadingState->roughness) * shadingState->opacity;

  // GGX microfacet distribution
  const float D = (alpha * alpha * heaviside(NdotH))
      / (float(M_PI) * pow2(NdotH * NdotH * (alpha * alpha - 1.f) + 1.f));

  // Masking-shadowing term
  const float G =
      ((2.f * fabsf(NdotL) * heaviside(LdotH))
          / (fabsf(NdotL)
              + sqrtf(alpha * alpha + (1.f - alpha * alpha) * NdotL * NdotL)))
      * ((2.f * fabsf(NdotV) * heaviside(VdotH))
          / (fabsf(NdotV)
              + sqrtf(alpha * alpha + (1.f - alpha * alpha) * NdotV * NdotV)));

  const float denom = 4.f * fabsf(NdotV) * fabsf(NdotL);
  const vec3 specularBRDF = denom != 0.f ? (F * D * G) / denom : vec3(0.f);

  return (diffuseBRDF + specularBRDF) * lightSample->radiance;
}
