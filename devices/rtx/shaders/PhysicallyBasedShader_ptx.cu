#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2025 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice,
 * this list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
 * LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
 * CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
 * SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
 * INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
 * CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
 * ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 * POSSIBILITY OF SUCH DAMAGE.
 */

#include "gpu/gpu_decl.h"
#include "gpu/gpu_objects.h"
#include "gpu/intersectRay.h"
#include "gpu/sampleLight.h"
#include "gpu/shadingState.h"
#include "gpu/shading_api.h"
#include "shaders/PhysicallyBasedShader.h"

using namespace visrtx;

// Signature must match the call inside shaderPhysicallyBasedSurface in
// PhysicallyBasedShader.cuh.
VISRTX_CALLABLE vec3 __direct_callable__evalSurfaceMaterial(
    const PhysicallyBasedShadingState *shadingState,
    const SurfaceHit *hit,
    const LightSample *lightSample,
    const vec3 *outgoingDir)
{
  const vec3 H = normalize(lightSample->dir + *outgoingDir);
  const float NdotH = dot(hit->Ns, H);
  const float NdotL = dot(hit->Ns, lightSample->dir);
  const float NdotV = dot(hit->Ns, *outgoingDir);
  const float VdotH = dot(*outgoingDir, H);
  const float LdotH = dot(lightSample->dir, H);

  // Fresnel
  const vec3 f0 = glm::mix(
      vec3(pow2((1.f - shadingState->ior) / (1.f + shadingState->ior))),
      shadingState->baseColor,
      shadingState->metallic);
  const vec3 F = f0 + (vec3(1.f) - f0) * pow5(1.f - fabsf(VdotH));

  // Metallic materials don't reflect diffusely:
  const vec3 diffuseColor =
      glm::mix(shadingState->baseColor, vec3(0.f), shadingState->metallic);

  const vec3 diffuseBRDF =
      (vec3(1.f) - F) * float(M_1_PI) * diffuseColor * fmaxf(0.f, NdotL);

  // Alpha
  const float alpha = pow2(shadingState->roughness) * shadingState->opacity;

  // GGX microfacet distribution
  const float D = (alpha * alpha * heaviside(NdotH))
      / (float(M_PI) * pow2(NdotH * NdotH * (alpha * alpha - 1.f) + 1.f));

  // Masking-shadowing term
  const float G =
      ((2.f * fabsf(NdotL) * heaviside(LdotH))
          / (fabsf(NdotL)
              + sqrtf(alpha * alpha + (1.f - alpha * alpha) * NdotL * NdotL)))
      * ((2.f * fabsf(NdotV) * heaviside(VdotH))
          / (fabsf(NdotV)
              + sqrtf(alpha * alpha + (1.f - alpha * alpha) * NdotV * NdotV)));

  const float denom = 4.f * fabsf(NdotV) * fabsf(NdotL);
  const vec3 specularBRDF = denom != 0.f ? (F * D * G) / denom : vec3(0.f);

  return (diffuseBRDF + specularBRDF) * lightSample->radiance;
}
